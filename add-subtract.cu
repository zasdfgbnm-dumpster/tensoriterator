#include <ATen/native/cuda/Loops.cuh>
#include <helper.cuh>

std::vector<int64_t> shape = {
  2, 3, 5
};
std::vector<std::vector<int64_t>> strides = {
  // warning: strides are in bytes!
  {4, 8, 24},
  {4, 8, 24},
  {4, 8, 24},
  {4, 8, 24},
};
std::vector<char *> data_ptrs = {
  nullptr, nullptr, nullptr, nullptr
};
bool is_contiguous = true;
int64_t noutputs = 2;
int64_t N = 5;

using namespace at;
using namespace at::native;

void compute() {
  std::cout << "is_contiguous = " << is_contiguous << std::endl;
  data_ptrs[0] = (char *)zeros<float>(N);
  data_ptrs[1] = (char *)zeros<float>(N);
  TensorIteratorBase iter;  // uses the hardcoded globals above
  gpu_kernel_multiple_outputs(iter, [] C10_HOST_DEVICE (float a, float b) {
    return thrust::tuple<float, float>(a + b, a - b);
  });
  hipDeviceSynchronize();
  print((float *)data_ptrs[0], N);
  print((float *)data_ptrs[1], N);
  std::cout << std::endl;
}

int main() {
  data_ptrs[2] = (char *)arange<float>(N);
  data_ptrs[3] = (char *)arange<float>(N);
  print((float *)data_ptrs[2], N);
  print((float *)data_ptrs[3], N);
  std::cout << std::endl;

  is_contiguous = true;
  compute();

  is_contiguous = false;
  compute();
}
