
#include <hip/hip_runtime.h>
#include <iostream>

constexpr int64_t N = 5;
constexpr int MAX_DIMS = 25;
__managed__ float data[N];


#define CHECK() do { auto code = hipGetLastError(); if(code != hipSuccess) throw std::runtime_error(hipGetErrorString(code)); } while(0)

struct useless {};

template<typename type, typename whatever>
struct container_base {
  type object;
  __device__ container_base(type obj, whatever unused): object(obj) {}
};

template <typename type>
struct container_derived : container_base<type, useless> {
  __device__ container_derived(type obj):
    container_base<type, useless>(obj, useless()) {}
};

struct OffsetCalculator {
  OffsetCalculator() : dims(3) {}

  __device__ int get(int i) const {
    int x = 0;

    #pragma unroll
    for (int dim = 0; dim < MAX_DIMS; ++dim) {
      if (dim == dims) {
        break;
      }
      x = i;
    }
    return x;
  }

  int dims;
  int whatever[MAX_DIMS * 2];
};


template <typename out_calc_t>
__global__ void range_kernel(float *data, out_calc_t oc) {
#ifdef BUG
  auto container = container_derived<out_calc_t>(oc);
  int offsets = container.object.get(blockIdx.x);
#else
  int offsets = oc.get(blockIdx.x);
#endif
  *(data + offsets) = blockIdx.x;
}

int main() {
  auto oc = OffsetCalculator();
  range_kernel<<<N, 1>>>(data, oc);
  hipDeviceSynchronize();
  CHECK();
  for (int64_t i = 0; i < N; i++) {
    std::cout << data[i] << ", ";
  }
  std::cout << std::endl;
}
