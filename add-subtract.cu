
#include <hip/hip_runtime.h>
#include <iostream>

constexpr int64_t N = 5;
__managed__ float data[N];


#define CHECK() do { auto code = hipGetLastError(); if(code != hipSuccess) throw std::runtime_error(hipGetErrorString(code)); } while(0)

struct useless {};

template<typename type, typename whatever>
struct container_base {
  type object;
  __device__ container_base(type obj, whatever unused): object(obj) {}
};

template <typename type>
struct container_derived : container_base<type, useless> {
  __device__ container_derived(type obj):
    container_base<type, useless>(obj, useless()) {}
};

struct echo {
  echo(): n(3) {}

  __device__ int get(int i) const {
    // this function just returns i
    int x = 0;

    if (n == 0) {
      return x;
    }
    x = i;

    return x;
  }

  int n;
  int whatever[50];
};


__global__ void range_kernel(float *data, echo obj) {
#ifdef BUG
  auto container = container_derived<echo>(obj);
  int offsets = container.object.get(blockIdx.x);
#else
  int offsets = obj.get(blockIdx.x);
#endif
  *(data + offsets) = blockIdx.x;
}

int main() {
  auto oc = echo();
  range_kernel<<<N, 1>>>(data, oc);
  hipDeviceSynchronize();
  CHECK();
  for (int64_t i = 0; i < N; i++) {
    std::cout << data[i] << ", ";
  }
  std::cout << std::endl;
}
