
#include <hip/hip_runtime.h>
#include <iostream>

constexpr int64_t N = 5;
__managed__ float data[N];

#define CHECK() do { auto code = hipGetLastError(); if(code != hipSuccess) throw std::runtime_error(hipGetErrorString(code)); } while(0)

struct echo {
  int n = 3;
  int whatever[50];  // the size of this array must be large enough

  __device__ int get(int i) const {
    // this function just returns i
    int x = 0;

    if (n == 0) {
      return x;
    }
    x = i;

    return x;
  }
};

struct useless {};

template<typename type>
struct base {
  type object;
  __device__ base(type obj, useless unused): object(obj) {}
};

template <typename type>
struct derived : base<type> {
  __device__ derived(type obj):
    base<type>(obj, useless()) {}
};


__global__ void range_kernel(float *data, echo obj) {
#ifdef BUG
  auto container = derived<echo>(obj);
  int offsets = container.object.get(blockIdx.x);
#else
  int offsets = obj.get(blockIdx.x);
#endif
  *(data + offsets) = blockIdx.x;
}

int main() {
  auto oc = echo();
  range_kernel<<<N, 1>>>(data, oc);
  hipDeviceSynchronize();
  CHECK();
  for (int64_t i = 0; i < N; i++) {
    std::cout << data[i] << ", ";
  }
  std::cout << std::endl;
}
