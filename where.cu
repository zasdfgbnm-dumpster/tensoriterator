
#include <hip/hip_runtime.h>
#include <iostream>
#include <tuple>

struct alignas(16) A {
  double data[2];
};

template<int arg_index>
struct initialize {
  template <typename args_t>
  static __device__ void apply(args_t *args, int j) {
    printf("%d%d\n", 0, 0);
    std::get<arg_index>(args[j]) = {};
  }
};

struct initialize_all {
  template<typename... Args>
  static inline __host__ __device__ void with_args(Args&&... args) {
    initialize<0>::apply(std::forward<Args>(args)...);
    initialize<1>::apply(std::forward<Args>(args)...);
  }
};

__global__ void unrolled_elementwise_kernel(A *result)
{
  std::tuple<bool, A> args[2];
  #pragma unroll
  for (int i = 0; i < 2; i++) {
    initialize_all::with_args(args, i);
  }

  if ((int)blockIdx.x >= 0) {
    return;
  }
  // code below will not be executed
  *result = std::get<1>(args[0]);
}

int main() {
  unrolled_elementwise_kernel<<<1, 1>>>(nullptr);
  hipDeviceSynchronize();
  auto code = hipGetLastError();
  if(code != hipSuccess) {
    std::string e = hipGetErrorString(code);
    std::cerr << e << std::endl;
    throw std::runtime_error(e);
  }
}
