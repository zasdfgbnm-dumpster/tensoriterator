
#include <hip/hip_runtime.h>
#include <iostream>
#include <tuple>

constexpr int thread_work_size = 4;

struct alignas(16) A {
  double data[2];
};

template<template<int i> typename func>
struct static_unroll {
  template<typename... Args>
  static inline __host__ __device__ void with_args(Args&&... args) {
    func<0>::apply(std::forward<Args>(args)...);
    func<1>::apply(std::forward<Args>(args)...);
    func<2>::apply(std::forward<Args>(args)...);
  }
};

// helper structs to be used with static_unroll to load arguments
// one by one

template<int arg_index>
struct unroll_load_helper {
  template <typename args_t>
  static __device__ void apply(args_t *args, int j) {
    uint64_t addr = 0;
    printf("address: %llu, mod: %llu\n", addr, addr % 16);
    std::get<arg_index>(args[j]) = {};
  }
};

__global__ void unrolled_elementwise_kernel(A *result, A *data)
{
  int idx = blockIdx.x;

  A results[4];
  std::tuple<bool, A, A> args[4];

  // load
  #pragma unroll
  for (int i = 0; i < thread_work_size; i++) {
    static_unroll<unroll_load_helper>::with_args(args, i);
  }

  if (idx >= 0) {
    return;
  }

  #pragma unroll
  for (int i = 0; i < 4; i++) {
    results[i] = std::get<1>(args[i]);
    *result = results[i];
  }
}

int main() {
  unrolled_elementwise_kernel<<<1, 1>>>(nullptr, nullptr);
  hipDeviceSynchronize();
  auto code = hipGetLastError();
  if(code != hipSuccess) {
    std::string e = hipGetErrorString(code);
    std::cerr << e << std::endl;
    throw std::runtime_error(e);
  }
}
