#include <ATen/native/cuda/Loops.cuh>
#include <helper.cuh>

std::vector<int64_t> shape = {
  5, 5, 5
};
std::vector<std::vector<int64_t>> strides = {
  // warning: strides are in bytes!
  {16, 80, 400},
  {1, 5, 25},
  {16, 80, 400},
  {0, 0, 0},
};
std::vector<at::ScalarType> dtypes = {
  at::ScalarType::ComplexDouble,
  at::ScalarType::Bool,
  at::ScalarType::ComplexDouble,
  at::ScalarType::ComplexDouble,
};
std::vector<char *> data_ptrs = {
  nullptr, nullptr, nullptr, nullptr,
};
bool is_contiguous = false;
int64_t noutputs = 1;

using namespace at;
using namespace at::native;

int main() {
  data_ptrs[0] = (char *)zeros<c10::complex<double>>(30);
  data_ptrs[1] = (char *)zeros<bool>(30);
  data_ptrs[2] = (char *)arange<c10::complex<double>>(30);
  data_ptrs[3] = (char *)arange<c10::complex<double>>(1);
  print((bool *)data_ptrs[1], 30);
  print((c10::complex<double> *)data_ptrs[2], 30);
  print((c10::complex<double> *)data_ptrs[3], 30);
  hipDeviceSynchronize();
  TensorIteratorBase iter;  // uses the hardcoded globals above
  gpu_kernel(iter, [] GPU_LAMBDA (bool cond, c10::complex<double> a, c10::complex<double> b) {
    return cond ? a : b;
  });
  hipDeviceSynchronize();
  print((c10::complex<double> *)data_ptrs[0], 30);
}
